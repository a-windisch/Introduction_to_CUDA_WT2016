#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "CUDA_function.h"
#include <vector>
#include <thrust/device_vector.h>

using namespace std;

__global__ void manipulate_matrix( int proc_id, int i_part_size, int dim, int work, double *devdata)
{

 int i = blockIdx.x*blockDim.x + threadIdx.x; //determine thread id i
 int j = blockIdx.y*blockDim.y + threadIdx.y; //determine thread id j

 int ip = proc_id*i_part_size + i; 
 int jp = j; 
 int pos = i*dim + j;
 for(int k=0; k<work; k++)
 {
  devdata[pos] += atan((double)((ip+k+1)%10)*acos(-1.0) + double((jp+k+1)%5)*acos(-1.0)*acos(-1.0));
 }
}


extern "C" vector<double> launch_CUDA_C_code( int num_procs, int proc_id, int dim, int work, vector<double> M_MPI_part) 
{

 hipSetDevice(proc_id);
 cout << "Process number " << proc_id << " claimed GPU number " <<  proc_id << ".\n";

 int i_part_size =dim/num_procs;
 vector<double> M_part(dim*i_part_size);
 thrust::host_vector<double> hostdata(dim*i_part_size);
 thrust::device_vector<double> devdata(dim*i_part_size);
 double * ptr = thrust::raw_pointer_cast(&devdata[0]);
 dim3 threadsPerBlock(16,16);         //number of threads per block and grid size
 int i_extent = i_part_size/threadsPerBlock.x;
 int j_extent = dim/threadsPerBlock.x;
 dim3 numBlocks(i_extent,j_extent);

 for( int i=0; i<dim*i_part_size; i++)
 {
  hostdata[i] = M_MPI_part[i];
 }
 
 thrust::copy(hostdata.begin(),hostdata.end(),devdata.begin());
 
 cout << "Process number " << proc_id << " calls the kernel. \n";

 manipulate_matrix<<<numBlocks,threadsPerBlock>>> ( proc_id, i_part_size, dim, work, ptr);
 hipDeviceSynchronize();
 cout << "Process number " << proc_id << " kernel call finished. \n";
 
 thrust::copy(devdata.begin(),devdata.end(),hostdata.begin());
 cout << "Process number " << proc_id << " data transfer to host complete. \n";
 
 for( int i=0; i<dim*i_part_size; i++)
 {
  M_part[i] = hostdata[i];
 }

 return M_part; 
}
