
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"

int  main ( void ) {

 using namespace std;


 int N = 7000;

 cout << "CUDA-C (cuBLAS) version\n";
 cout << "Initializing cuBLAS.\n";

 hipblasStatus_t status;
 cublasInit();

 cout << "Done.\n";
 cout << "Allocating " << N << "x" << N << "-matrices MA, MB and MC on host.\n";
 double *MA_h = (double*)malloc(N*N*sizeof(double));
 double *MB_h = (double*)malloc(N*N*sizeof(double));
 double *MC_h = (double*)malloc(N*N*sizeof(double));
 double *MA_d; 
 double *MB_d; 
 double *MC_d;

 cout << "Done.\n";
 cout << "Filling matrices on host.\n";

 for (int i=0;i<N;i++)
 {
  for (int j=0;j<N;j++)
  {
    int pos = j*N + i;
    MA_h[pos] = (double) ( i%10 + j%10 + 1);   
    MB_h[pos] = (double) ( i%8  + j%11 + 1); 
    MC_h[pos] = 0.0;
  }
 }

 cout << "Done.\n";
 cout << "Allocating " << N << "x" << N << "-matrices MA, MB and MC on device.\n";
 cout << "(in fact, they are N*N 1dim vectors here...)\n";


 //This can be done in a secure way, by monitoring the process
 //using the status variable defined above. In case anything goes
 //wrong, an error message will be produced.
 status = cublasAlloc(N*N, sizeof(*MA_d), (void**)&MA_d);
 if (status != HIPBLAS_STATUS_SUCCESS) {
     cout <<  "!!!! device memory allocation error (MA_d)\n";
     return EXIT_FAILURE;
 }

 //If you want to spice up your life by adding excitement
 //by NOT monitoring the status, that is perfectly fine as well.
 //Let us live on the edge for the rest of the code.
 cublasAlloc(N*N,sizeof(*MB_d),(void**) &MB_d);
 cublasAlloc(N*N,sizeof(*MC_d),(void**) &MC_d);

 cout << "Done.\n";
 cout << "Copy data from host to device.\n";

 hipblasSetVector(N*N,sizeof(double),MA_h,1,MA_d,1);
 hipblasSetVector(N*N,sizeof(double),MB_h,1,MB_d,1);

 cout << "Done.\n";
 cout << "Perform multiplication MC = MA.MB on device.\n";
 cublasDgemm('N','N',N,N,N,1.0,MA_d,N,MB_d,N,0.0,MC_d,N);
 cout << "Done.\n";
 cout << "Copy data from device to host.\n";

 hipblasGetVector(N*N,sizeof(double),MC_d,1,MC_h,1);


 cout << "Done.\n";
 cout << "Control element MC(123,456):\n";
 cout << MC_h[122+455*N]<<"\n";

 free(MA_h);  
 free(MB_h);  
 free(MC_h);
 cublasFree(MA_d);
 cublasFree(MB_d);
 cublasFree(MC_d);

 cublasShutdown();

}

