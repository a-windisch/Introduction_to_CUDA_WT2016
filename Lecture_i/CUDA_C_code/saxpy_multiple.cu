
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{

 using namespace std;

 int N=1<<20; //shift 20 bits to the left
 int num=100000; 
 float a=2.0;
 float *x;  //host array x
 float *y;  //host array y
 float *x_d;  //device array x
 float *y_d;  //device array y

 x = new (nothrow) float [N];
 y = new (nothrow) float [N];
 hipMalloc(&x_d, N*sizeof(float)); //allocate memory for x on device
 hipMalloc(&y_d, N*sizeof(float)); //allocate memory for y on device

 for (int i=0; i<N; i++ ) //fill host arrays 
 {
  x[i]=(float)i;
  y[i]=(float)2*i;
 }

 //transfer arrays to device
 hipMemcpy(x_d, x, N*sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(y_d, y, N*sizeof(float), hipMemcpyHostToDevice);

 cout <<"\n";
 cout <<"Performing "<<num<<" CUDA-C SAXPY on 2^20 elements.\n";

 for( int j=0; j<num; j++) 
 {
  saxpy<<<(N+255)/256, 256>>>(N, a, x_d, y_d); // Perform SAXPY on 1M elements
 }

 //transfer arrays to device
 hipMemcpy(y, y_d, N*sizeof(float), hipMemcpyDeviceToHost);

 cout <<"Done.\n";
 cout <<"y[213]="<<y[213]<<"\n";
 cout <<"\n";

 return 0;
}



