#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1024
#define START_SCALE 1.5f
#define MAXIT 256
#define C_RE -0.8f
#define C_IM 0.156f
#define ZOOM 200

//======================================================================
/*This function checks whether a point
 belongs to the filled julia set. It 
 returns 0 if the value 'escaped', and
 one if the maximal numer of iterations
 has been reached. It is a function on the 
 device, but it can't be called like a
 kernel from the host. It is used by the
 kernel 'construct_julia_set' below. Note that
 the kernel has the 'global' attribute rather
 than the 'device' attribute.*/
__device__ int julia( int i, int j, float scale)
{

 //rescale grid to actual scale
 float x = scale * (float)(SIZE/2 - i)/(SIZE/2);
 float y = scale * (float)(SIZE/2 - j)/(SIZE/2);

 //real and imaginary part of point in question
 float z_re=x;
 float z_im=y;
 float z_re_old;

 //compute Z(n+1) = Zn^2 + C
 for (int k=0; k<MAXIT;k++)
 {
  z_re_old = z_re;                              //store old real value
  z_re =(z_re*z_re-z_im*z_im) + (C_RE);         //compute Re(Z(n+1))
  z_im = 2.0f*z_re_old*z_im + (C_IM);           //compute Im(Z(n+1))
  if ( sqrt(z_re*z_re+z_im*z_im) > SIZE)        //check if point escaped
  {
   return 0;                                    //point escaped
  }
 }
 return 1;                                      //point in set
}

//======================================================================
/* This function uses the one defined above
 to construct an array called 'set' with ones and zeroes, 
 defining the elements of the julia set. This is the
 kernel function to be called from the host and exectued
 on the device. It thus carries the 'global' attribute.*/
__global__ void construct_julia_set( int *set, float scale)
{
// int i = blockIdx.x*blockDim.x + threadIdx.x;	//determine thread id i
// int j = blockIdx.y*blockDim.y + threadIdx.y;	//determine thread id j
 int i = blockIdx.x;
 int j = blockIdx.y;
 int pos = i + j * SIZE;			//remap to 1-dim array
 set[pos] = julia( i, j, scale);              	//fill the set
}
//======================================================================
int main( void )
{

 using namespace std;

 int set[SIZE*SIZE];                    //the result array on the host
 int *set_d; 		                //the result array on the device
 int written;                           //aux variable for writing file
 int num;                               //numbering of output file
 float x;                               //Re part in complex Z plane
 float y;                               //Im part in complez Z plane
 float scale;                           //initial scale for zoom
 char buffer[32];                       //buffer for filenames
 FILE *out;                             //output file                           
 //dim3 threadsPerBlock(16,16);		//number of threads per block and grid size
 //dim3 numBlocks(SIZE/threadsPerBlock.x,SIZE/threadsPerBlock.y);
 dim3 grid(SIZE,SIZE);

 for(int k=0; k<SIZE*SIZE; k++)
 {
  set[k] = 0;
 }
 hipMalloc(&set_d, SIZE*SIZE*sizeof(int)); //allocate memory for set on device

 /*The variable 'written' allows us to introduce a newline 
   after each row that has been written to the outpur file. 
   This allows for more freedom in printing the set using
   gnuplot.*/

 written=0;                                             //reset 'written' value
 for( int k=0; k<ZOOM; k++ )                             //k...number of zoom slices to produce
 {                                                      //vary scale for zoom
  scale = START_SCALE *(400.0f-(float)k)/400.0f + 0.01;
  hipMemcpy(set_d, set, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice); //init set on device
//  construct_julia_set<<<numBlocks,threadsPerBlock>>>(set_d, scale); //construct julia set on GPU
  construct_julia_set<<<grid,1>>>(set_d, scale); //construct julia set on GPU
  hipDeviceSynchronize();
  hipMemcpy(set, set_d, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost); //copy resultto host
  num = k;                                              //out: 'julia_000.dat', 'julia_001.dat',...
  snprintf(buffer, sizeof(char)*32, "julia_%03i.dat", num);
  out = fopen( buffer, "wt" );                          //write in text mode (wt)
  for (int i=0; i<SIZE; i++)                            //actual grid values x and y
  {
   x = scale * (float)(SIZE/2 - i)/(SIZE/2);
   for (int j=0; j<SIZE; j++)
   {
    y = scale * (float)(SIZE/2 - j)/(SIZE/2);
    int pos = i + j * SIZE;                             //position in  array
    if(set[pos]==1)                                     //write only if part of set
    {
     fprintf(out,"%f %f \n",x,y);
     written = 1;                                       //set written to 1
    }
   }//end inner grid loop (j)
   if( written == 1 )
   {
    fprintf(out,"\n");                                  //add newline if row content not empty
   }
   written=0;                                           //reset written value
  }//end outer grid loop (i)
 fclose(out);                                           //close file
 }//end zoom loop (k)
 hipFree( set_d );					//deallocate
}
